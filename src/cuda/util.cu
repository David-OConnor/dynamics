
#include <hip/hip_runtime.h>
// This module contains constants and utility functions related to the kernels we use.


// Allows easy switching between float and double.
using dtype = float;
using dtype3 = float3;


__device__
const float SOFTENING_FACTOR_SQ = 0.000001f;

__device__
const float TAU = 6.283185307179586f;

// 1/sqrt(pi)
__device__
// const float INV_SQRT_PI = 1.0f / sqrtf(CUDART_PI_F);
const float INV_SQRT_PI = 0.5641895835477563f;

// __device__
// const float EPS_DIV0 = 0.00000000001f;

// Vector operations for float3
__device__ inline float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ inline float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ inline float3 operator/(const float3 &a, const float b) {
    return make_float3(a.x / b, a.y / b, a.z / b);
}

__device__ inline float3 operator*(const float3 &a, const float b) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

extern "C" __global__ void zero_f32(float* __restrict__ x, unsigned int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) x[i] = 0.0f;
}

extern "C" __global__ void zero_f64(double* __restrict__ x, unsigned int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) x[i] = 0.0;
}

// For returning both from a function.
struct ForceEnergy {
    float3 force;
    float energy;
};

// Apparently normally adding to output can cause race conditions.
__device__ __forceinline__ void atomicAddFloat3(float3* addr, const float3 v) {
    atomicAdd(&addr->x, v.x);
    atomicAdd(&addr->y, v.y);
    atomicAdd(&addr->z, v.z);
}

__device__ inline float3 min_image(float3 ext, float3 dv) {
    dv.x -= rintf(dv.x / ext.x) * ext.x;
    dv.y -= rintf(dv.y / ext.y) * ext.y;
    dv.z -= rintf(dv.z / ext.z) * ext.z;

    return dv;
}

// These params includes inv_r due to it being shared with LJ.
__device__
ForceEnergy coulomb_force_spme_short_range(
    float r,
    float inv_r,
    float3 dir,
    float q_0,
    float q_1,
    float cutoff_dist,
    float alpha
) {
    ForceEnergy result;

    // Outside cutoff: no short-range contribution
    if (r >= cutoff_dist) {
        result.force  = make_float3(0.f, 0.f, 0.f);
        result.energy = 0.f;
        return result;
    }

    const float alpha_r = alpha * r;
    const float erfc_term = erfcf(alpha_r);
    const float charge_term = q_0 * q_1;

    const float exp_term  = __expf(-(alpha_r * alpha_r));

    const float inv_r_sq = inv_r * inv_r;
    const float coef = 2.0f * alpha * exp_term * INV_SQRT_PI;
    // mul_add
    const float force_mag = charge_term * fmaf(erfc_term, inv_r_sq, coef * inv_r);

    result.force = dir * force_mag;
    result.energy = charge_term * inv_r * erfc_term;
    return result;
}

// This assumes diff (and dir) is in order tgt - src.
// Different API.
__device__
ForceEnergy lj_force_v2(
    float3 diff,
    float r,
    float inv_r,
    float3 dir,
    float sigma,
    float eps
) {
    const float sr = sigma * inv_r;
    const float sr2 = sr * sr;
    const float sr4 = sr2 * sr2;
    const float sr6 = sr4 * sr2;
    const float sr12 = sr6 * sr6;

    // Optimized mul_add.
    const float mag = 24.0f * eps * fmaf(2.f, sr12, -sr6) * inv_r;

    ForceEnergy result;
    result.force = dir * mag;
    result.energy = 4.f * eps * (sr12 - sr6);

    return result;
}

// Helpers to unflatten positions
// todo: QC these.
__device__ inline float3 ld3(const float* a, uint32_t i) {
    // a is flattened [x0 y0 z0 x1 y1 z1 ...], i = atom index
    const uint32_t j = 3u * i;
    return make_float3(a[j+0], a[j+1], a[j+2]);
}

__device__ inline float3 load_pos(
    uint8_t atom_type, uint8_t water_site, uint32_t idx,
    const float* pos_dyn,
    const float* pos_w_o, const float* pos_w_m,
    const float* pos_w_h0, const float* pos_w_h1
){
    if (atom_type == 0) return ld3(pos_dyn, idx);
    // water
    switch (water_site) {
        case 1: return ld3(pos_w_o,  idx);
        case 2: return ld3(pos_w_m,  idx);
        case 3: return ld3(pos_w_h0, idx);
        case 4: return ld3(pos_w_h1, idx);
        default: return make_float3(0.f,0.f,0.f); // shouldn't happen
    }
}

// Helpers to avoid atomic adds on energies
__inline__ __device__ double warp_sum(double x) {
    for (int offset = 16; offset > 0; offset >>= 1)
        x += __shfl_down_sync(0xffffffff, x, offset);
    return x;
}

__inline__ __device__ double block_sum(double x) {
    static __shared__ double smem[32]; // one lane-0 per warp
    int lane = threadIdx.x & 31;
    int wid  = threadIdx.x >> 5;

    x = warp_sum(x);
    if (lane == 0) smem[wid] = x;
    __syncthreads();

    x = (threadIdx.x < blockDim.x / 32) ? smem[lane] : 0.0;
    if (wid == 0) x = warp_sum(x);
    return x;
}
